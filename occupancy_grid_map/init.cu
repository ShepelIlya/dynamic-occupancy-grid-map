#include "hip/hip_runtime.h"
#include "occupancy_grid_map.h"
#include "common.h"
#include "cuda_utils.h"

#include <thrust/random.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void initParticlesKernel(Particle* particle_array, int width, int height, int size)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ARRAY_SIZE(particle_array); i += blockDim.x * gridDim.x)
	{
		int size = width * height;

		thrust::default_random_engine rng;
		thrust::uniform_int_distribution<int> dist_idx(0, size);
		thrust::normal_distribution<float> dist_vel(0.0f, 4.0f);

		int index = dist_idx(rng);

		float x = index % width + 0.5f;
		float y = index / width + 0.5f;

		particle_array[i].weight = 1.0f / size;
		//particle_array[i].state << x, y, dist_vel(rng), dist_vel(rng);
	}
}

void OccupancyGridMap::initialize()
{
	initParticlesKernel<<<divUp(ARRAY_SIZE(particle_array), 256), 256>>>(particle_array, params.width, params.height,
		ARRAY_SIZE(particle_array));

	CHECK_ERROR(hipGetLastError());
}
