#include "occupancy_grid_map.h"
#include "cuda_utils.h"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/random.h>
#include <Eigen/Dense>

#include "hip/hip_runtime.h"

OccupancyGridMap::OccupancyGridMap(const GridParams& params)
	: params(params)
{
	int c = (params.width / params.resolution) * (params.height / params.resolution);

	hipMallocManaged((void**)&grid_cell_array, c * sizeof(GridCell));
	hipMallocManaged((void**)&particle_array, params.v * sizeof(Particle));
	hipMallocManaged(&weight_array, params.v * sizeof(float));
	hipMallocManaged(&birth_weight_array, params.v * sizeof(float));
	hipMallocManaged(&meas_cell_array, c * sizeof(MeasurementCell));

	hipMalloc(&born_masses_array, c * sizeof(float));
	hipMalloc(&vel_x_array, params.v * sizeof(float));
	hipMalloc(&vel_y_array, params.v * sizeof(float));
	hipMalloc(&vel_x_squared_array, params.v * sizeof(float));
	hipMalloc(&vel_y_squared_array, params.v * sizeof(float));
	hipMalloc(&vel_xy_array, params.v * sizeof(float));
	hipMalloc(&rand_array, params.v * sizeof(float));

	initialize();
}

OccupancyGridMap::~OccupancyGridMap()
{
	hipFree(grid_cell_array);
	hipFree(particle_array);
	hipFree(weight_array);
	hipFree(birth_weight_array);
	hipFree(meas_cell_array);

	hipFree(born_masses_array);
	hipFree(vel_x_array);
	hipFree(vel_y_array);
	hipFree(vel_x_squared_array);
	hipFree(vel_y_squared_array);
	hipFree(vel_xy_array);
	hipFree(rand_array);
}

void OccupancyGridMap::update(float dt)
{
	particlePrediction(dt);
	particleAssignment();
	gridCellOccupancyUpdate();
	updatePersistentParticles();
	initializeNewParticles();
	statisticalMoments();
	resampling();
}
