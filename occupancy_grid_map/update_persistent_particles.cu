#include "hip/hip_runtime.h"
#include "occupancy_grid_map.h"
#include "common.h"
#include "cuda_utils.h"

#include <hip/hip_runtime.h>
#include <>

__device__ float update_unnorm(Particle* particle_array, int i, MeasurementCell* meas_cell_array)
{
	Particle& particle = particle_array[i];
	return meas_cell_array[particle.grid_cell_idx].likelihood * particle.weight;
}

__device__ float calc_norm_assoc(float occAccum, float rhoP)
{
	return rhoP / occAccum;
}

__device__ float calc_norm_unassoc(const GridCell& gridCell)
{
	return gridCell.pers_occ_mass / gridCell.occ_mass;
}

__device__ void set_normalization_components(GridCell* grid_cell_array, int i, float mu_A, float mu_UA)
{
	grid_cell_array[i].mu_A = mu_A;
	grid_cell_array[i].mu_UA = mu_UA;
}

__device__ float normalize(Particle& particle, GridCell* grid_cell_array, MeasurementCell* meas_cell_array, float weight)
{
	GridCell& cell = grid_cell_array[particle.grid_cell_idx];
	MeasurementCell& measCell = meas_cell_array[particle.grid_cell_idx];

	return measCell.p_A * cell.mu_A * weight + (1.0f - measCell.p_A) * cell.mu_UA * particle.weight;
}

__global__ void updatePersistentParticlesKernel1(Particle* particle_array, MeasurementCell* meas_cell_array, float* weight_array)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ARRAY_SIZE(particle_array); i += blockDim.x * gridDim.x)
	{
		weight_array[i] = update_unnorm(particle_array, i, meas_cell_array);
	}
}

__global__ void updatePersistentParticlesKernel2(GridCell* grid_cell_array, float* weight_array_accum)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ARRAY_SIZE(grid_cell_array); i += blockDim.x * gridDim.x)
	{
		int start_idx = grid_cell_array[i].start_idx;
		int end_idx = grid_cell_array[i].end_idx;
		float occ_accum = subtract(weight_array_accum, start_idx, end_idx);
		float rho_p = grid_cell_array[i].pers_occ_mass;
		float mu_A = calc_norm_assoc(occ_accum, rho_p);
		float mu_UA = calc_norm_unassoc(grid_cell_array[i]);
		set_normalization_components(grid_cell_array, i, mu_A, mu_UA);
	}
}

__global__ void updatePersistentParticlesKernel3(Particle* particle_array, MeasurementCell* meas_cell_array, GridCell* grid_cell_array,
	float* weight_array)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ARRAY_SIZE(particle_array); i += blockDim.x * gridDim.x)
	{
		weight_array[i] = normalize(particle_array[i], grid_cell_array, meas_cell_array, weight_array[i]);
	}
}

void OccupancyGridMap::updatePersistentParticles()
{
	updatePersistentParticlesKernel1<<<divUp(ARRAY_SIZE(particle_array), 256), 256>>>(particle_array, meas_cell_array, weight_array);

	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());

	thrust::device_vector<float> weightsAccum = accumulate(weight_array);
	float* weight_array_accum = thrust::raw_pointer_cast(weightsAccum.data());

	updatePersistentParticlesKernel2<<<divUp(ARRAY_SIZE(grid_cell_array), 256), 256>>>(grid_cell_array, weight_array_accum);

	CHECK_ERROR(hipGetLastError());

	updatePersistentParticlesKernel3<<<divUp(ARRAY_SIZE(particle_array), 256), 256>>>(particle_array, meas_cell_array,
		grid_cell_array, weight_array);

	CHECK_ERROR(hipGetLastError());
}
