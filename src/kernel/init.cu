#include "hip/hip_runtime.h"
#include "kernel/init.h"
#include "common.h"
#include "cuda_utils.h"

#include <thrust/random.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void initParticlesKernel(Particle* particle_array, int width, int height, int particles_size)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ARRAY_SIZE(particle_array); i += blockDim.x * gridDim.x)
	{
		thrust::default_random_engine rng;
		thrust::uniform_int_distribution<int> dist_idx(0, width * height);
		thrust::normal_distribution<float> dist_vel(0.0f, 4.0f);

		int index = dist_idx(rng);

		float x = index % width + 0.5f;
		float y = index / width + 0.5f;

		particle_array[i].weight = 1.0f / particles_size;
		particle_array[i].state = glm::vec4(x, y, dist_vel(rng), dist_vel(rng));
	}
}
