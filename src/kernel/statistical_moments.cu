#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael K�sel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include "kernel/statistical_moments.h"
#include "common.h"
#include "cuda_utils.h"

#include <hip/hip_runtime.h>
#include <>

__device__ float calc_mean(float* vel_array_accum, int start_idx, int end_idx, float rho_p)
{
	if (rho_p > 0.0f)
	{
		float vel_accum = subtract(vel_array_accum, start_idx, end_idx);
		return (1.0f / rho_p) * vel_accum;
	}
	return 0.0f;
}

__device__ float calc_variance(float* vel_squared_array_accum, int start_idx, int end_idx, float rho_p, float mean_vel)
{
	if (rho_p > 0.0f)
	{
		float vel_accum = subtract(vel_squared_array_accum, start_idx, end_idx);
		return (1.0f / rho_p) * vel_accum - mean_vel * mean_vel;
	}
	return 0.0f;
}

__device__ float calc_covariance(float* vel_xy_array_accum, int start_idx, int end_idx, float rho_p, float mean_x_vel, float mean_y_vel)
{
	if (rho_p > 0.0f)
	{
		float vel_accum = subtract(vel_xy_array_accum, start_idx, end_idx);
		return (1.0f / rho_p) * vel_accum - mean_x_vel * mean_y_vel;
	}
	return 0.0f;
}

__device__ void store(GridCell* grid_cell_array, int j, float mean_x_vel, float mean_y_vel, float var_x_vel, float var_y_vel,
	float covar_xy_vel)
{
	grid_cell_array[j].mean_x_vel = mean_x_vel;
	grid_cell_array[j].mean_y_vel = mean_y_vel;
	grid_cell_array[j].var_x_vel = var_x_vel;
	grid_cell_array[j].var_y_vel = var_y_vel;
	grid_cell_array[j].covar_xy_vel = covar_xy_vel;
}

__global__ void statisticalMomentsKernel1(Particle* particle_array, float* weight_array, float* vel_x_array, float* vel_y_array,
	float* vel_x_squared_array, float* vel_y_squared_array, float* vel_xy_array, int particle_count)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < particle_count)
	{
		float weight = weight_array[i];
		float vel_x = particle_array[i].state[2];
		float vel_y = particle_array[i].state[3];
		vel_x_array[i] = weight * vel_x;
		vel_y_array[i] = weight * vel_y;
		vel_x_squared_array[i] = weight * vel_x * vel_x;
		vel_y_squared_array[i] = weight * vel_y * vel_y;
		vel_xy_array[i] = weight * vel_x * vel_y;

		//printf("vx: %f, vy: %f\n", vel_x_array[i], vel_y_array[i]);
	}
}

__global__ void statisticalMomentsKernel2(GridCell* grid_cell_array, float* vel_x_array_accum, float* vel_y_array_accum,
	float* vel_x_squared_array_accum, float* vel_y_squared_array_accum, float* vel_xy_array_accum, int cell_count)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < cell_count)
	{
		float rho_p = grid_cell_array[i].pers_occ_mass;
		//printf("rho p: %f\n", rho_p);

		int start_idx = grid_cell_array[i].start_idx;
		int end_idx = grid_cell_array[i].end_idx;
		float mean_x_vel = calc_mean(vel_x_array_accum, start_idx, end_idx, rho_p);
		float mean_y_vel = calc_mean(vel_y_array_accum, start_idx, end_idx, rho_p);
		float var_x_vel = calc_variance(vel_x_squared_array_accum, start_idx, end_idx, rho_p, mean_x_vel);
		float var_y_vel = calc_variance(vel_y_squared_array_accum, start_idx, end_idx, rho_p, mean_y_vel);
		float covar_xy_vel = calc_covariance(vel_xy_array_accum, start_idx, end_idx, rho_p, mean_x_vel, mean_y_vel);

		//printf("x: %f, y: %f\n", mean_x_vel, mean_y_vel);

		store(grid_cell_array, i, mean_x_vel, mean_y_vel, var_x_vel, var_y_vel, covar_xy_vel);
	}
}
