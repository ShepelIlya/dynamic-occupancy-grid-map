#include "hip/hip_runtime.h"
#include "occupancy_grid_map.h"
#include "cuda_utils.h"
#include "common.h"

#include "kernel/measurement_grid.h"
#include "kernel/init.h"
#include "kernel/predict.h"
#include "kernel/particle_to_grid.h"
#include "kernel/mass_update.h"
#include "kernel/init_new_particles.h"
#include "kernel/update_persistent_particles.h"
#include "kernel/statistical_moments.h"
#include "kernel/resampling.h"

#include "opengl/renderer.h"
#include "opengl/texture.h"
#include "opengl/framebuffer.h"

#include <thrust/device_ptr.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <hip/hip_runtime.h>

int OccupancyGridMap::BLOCK_SIZE = 256;

OccupancyGridMap::OccupancyGridMap(const GridParams& params, const LaserSensorParams& laser_params)
	: params(params),
	  laser_params(laser_params),
	  grid_width(static_cast<int>(params.width / params.resolution)),
	  grid_height(static_cast<int>(params.height / params.resolution)),
	  particle_count(params.particle_count),
	  grid_cell_count(grid_width * grid_height),
	  new_born_particle_count(params.new_born_particle_count)
{
	CHECK_ERROR(hipMallocManaged((void**)&grid_cell_array, grid_cell_count * sizeof(GridCell)));
	CHECK_ERROR(hipMallocManaged((void**)&particle_array, particle_count * sizeof(Particle)));
	CHECK_ERROR(hipMallocManaged((void**)&particle_array_next, particle_count * sizeof(Particle)));
	CHECK_ERROR(hipMalloc((void**)&birth_particle_array, new_born_particle_count * sizeof(Particle)));

	CHECK_ERROR(hipMallocManaged((void**)&meas_cell_array, grid_cell_count * sizeof(MeasurementCell)));

	CHECK_ERROR(hipMalloc(&weight_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&birth_weight_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&born_masses_array, grid_cell_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_x_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_y_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_x_squared_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_y_squared_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_xy_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&rand_array, particle_count * sizeof(float)));

	initialize();
}

OccupancyGridMap::~OccupancyGridMap()
{
	CHECK_ERROR(hipFree(grid_cell_array));
	CHECK_ERROR(hipFree(particle_array));
	CHECK_ERROR(hipFree(particle_array_next));
	CHECK_ERROR(hipFree(meas_cell_array));

	CHECK_ERROR(hipFree(weight_array));
	CHECK_ERROR(hipFree(birth_weight_array));
	CHECK_ERROR(hipFree(born_masses_array));
	CHECK_ERROR(hipFree(vel_x_array));
	CHECK_ERROR(hipFree(vel_y_array));
	CHECK_ERROR(hipFree(vel_x_squared_array));
	CHECK_ERROR(hipFree(vel_y_squared_array));
	CHECK_ERROR(hipFree(vel_xy_array));
	CHECK_ERROR(hipFree(rand_array));
	
	delete renderer;
}

void OccupancyGridMap::initialize()
{
	initParticlesKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, grid_width, grid_height,
		particle_count);

	initGridCellsKernel<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(grid_cell_array, grid_width, grid_height,
		grid_cell_count);

	CHECK_ERROR(hipGetLastError());
	
	renderer = new Renderer(grid_width, grid_height, laser_params.fov);
}

void OccupancyGridMap::updateDynamicGrid(float dt)
{
	particlePrediction(dt);
	particleAssignment();
	gridCellOccupancyUpdate();
	updatePersistentParticles();
	initializeNewParticles();
	statisticalMoments();
	resampling();

	CHECK_ERROR(hipMemcpy(particle_array, particle_array_next, particle_count * sizeof(Particle), hipMemcpyDeviceToDevice));

	CHECK_ERROR(hipDeviceSynchronize());
}

void OccupancyGridMap::updateMeasurementGrid(float* measurements, int num_measurements)
{
	std::cout << "OccupancyGridMap::updateMeasurementGrid" << std::endl;

	float* d_measurements;
	CHECK_ERROR(hipMalloc(&d_measurements, num_measurements * sizeof(float)));
	CHECK_ERROR(hipMemcpy(d_measurements, measurements, num_measurements * sizeof(float), hipMemcpyHostToDevice));

	const int polar_width = num_measurements;
	const int polar_height = grid_height;

	dim3 block_dim(32, 32);
	dim3 grid_dim(divUp(polar_width, block_dim.x), divUp(polar_height, block_dim.y));
	dim3 cart_grid_dim(divUp(grid_width, block_dim.x), divUp(grid_height, block_dim.y));

	const float anisotropy_level = 16.0f;
	Texture texture(polar_width, polar_height, anisotropy_level);
	hipSurfaceObject_t polar_surface;
	
	// create polar texture
	texture.beginCudaAccess(&polar_surface);
	createPolarGridTextureKernel<<<grid_dim, block_dim>>>(polar_surface, d_measurements, polar_width, polar_height, params.resolution);

	CHECK_ERROR(hipGetLastError());
	texture.endCudaAccess(polar_surface);
	
	// render cartesian image to texture using polar texture
	renderer->renderToTexture(texture);
	
	Framebuffer* framebuffer = renderer->getFrameBuffer();
	hipSurfaceObject_t cartesian_surface;

	framebuffer->beginCudaAccess(&cartesian_surface);
	// transform RGBA texture to measurement grid
	cartesianGridToMeasurementGridKernel<<<cart_grid_dim, block_dim>>>(meas_cell_array, cartesian_surface, grid_width, grid_height);

	CHECK_ERROR(hipGetLastError());
	framebuffer->endCudaAccess(cartesian_surface);

	CHECK_ERROR(hipFree(d_measurements));
	CHECK_ERROR(hipDeviceSynchronize());
}

void OccupancyGridMap::particlePrediction(float dt)
{
	std::cout << "OccupancyGridMap::particlePrediction" << std::endl;

	glm::mat4x4 transition_matrix(1, 0, dt, 0, 
								  0, 1, 0, dt, 
								  0, 0, 1, 0, 
								  0, 0, 0, 1);

	thrust::default_random_engine rng;
	thrust::normal_distribution<float> dist_pos(0.0f, params.process_noise_position);
	thrust::normal_distribution<float> dist_vel(0.0f, params.process_noise_velocity);

	glm::vec4 process_noise(dist_pos(rng), dist_pos(rng), dist_vel(rng), dist_vel(rng));

	predictKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, grid_width, grid_height, params.p_S,
		transition_matrix, process_noise, particle_count);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::particleAssignment()
{
	std::cout << "OccupancyGridMap::particleAssignment" << std::endl;

	CHECK_ERROR(hipDeviceSynchronize());
	thrust::device_ptr<Particle> particles(particle_array);
	thrust::sort(particles, particles + particle_count, GPU_LAMBDA(Particle x, Particle y)
	{
		return x.grid_cell_idx < y.grid_cell_idx;
	});

	particleToGridKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, grid_cell_array, weight_array,
		particle_count);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::gridCellOccupancyUpdate()
{
	std::cout << "OccupancyGridMap::gridCellOccupancyUpdate" << std::endl;

	CHECK_ERROR(hipDeviceSynchronize());
	thrust::device_vector<float> weightsAccum(particle_count);
	accumulate(weight_array, weightsAccum);
	float* weight_array_accum = thrust::raw_pointer_cast(weightsAccum.data());

	gridCellPredictionUpdateKernel<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(grid_cell_array, particle_array, weight_array_accum,
		meas_cell_array, born_masses_array, params.p_B, params.p_S, grid_cell_count);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::updatePersistentParticles()
{
	std::cout << "OccupancyGridMap::updatePersistentParticles" << std::endl;

	updatePersistentParticlesKernel1<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, meas_cell_array,
		weight_array, particle_count);

	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());

	thrust::device_vector<float> weightsAccum(particle_count);
	accumulate(weight_array, weightsAccum);
	float* weight_array_accum = thrust::raw_pointer_cast(weightsAccum.data());

	updatePersistentParticlesKernel2<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(grid_cell_array,
		weight_array_accum, grid_cell_count);

	CHECK_ERROR(hipGetLastError());

	updatePersistentParticlesKernel3<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, meas_cell_array,
		grid_cell_array, weight_array, particle_count);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::initializeNewParticles()
{
	std::cout << "OccupancyGridMap::initializeNewParticles" << std::endl;

	CHECK_ERROR(hipDeviceSynchronize());

	thrust::device_vector<float> particleOrdersAccum(grid_cell_count);
	accumulate(born_masses_array, particleOrdersAccum);
	float* particle_orders_array_accum = thrust::raw_pointer_cast(particleOrdersAccum.data());

	normalize_particle_orders(particle_orders_array_accum, grid_cell_count, params.new_born_particle_count);

	initNewParticlesKernel1<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, grid_cell_array,
		meas_cell_array, weight_array, born_masses_array, birth_particle_array, particle_orders_array_accum, grid_cell_count);

	CHECK_ERROR(hipGetLastError());

	initNewParticlesKernel2<<<divUp(new_born_particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(birth_particle_array,
		grid_cell_array, birth_weight_array, grid_width, new_born_particle_count);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::statisticalMoments()
{
	std::cout << "OccupancyGridMap::statisticalMoments" << std::endl;

	statisticalMomentsKernel1<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, weight_array,
		vel_x_array, vel_y_array, vel_x_squared_array, vel_y_squared_array, vel_xy_array, particle_count);

	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());

	thrust::device_vector<float> velXAccum(particle_count);
	accumulate(vel_x_array, velXAccum);
	float* vel_x_array_accum = thrust::raw_pointer_cast(velXAccum.data());

	thrust::device_vector<float> velYAccum(particle_count);
	accumulate(vel_y_array, velYAccum);
	float* vel_y_array_accum = thrust::raw_pointer_cast(velYAccum.data());

	thrust::device_vector<float> velXSquaredAccum(particle_count);
	accumulate(vel_x_squared_array, velXSquaredAccum);
	float* vel_x_squared_array_accum = thrust::raw_pointer_cast(velXSquaredAccum.data());

	thrust::device_vector<float> velYSquaredAccum(particle_count);
	accumulate(vel_y_squared_array, velYSquaredAccum);
	float* vel_y_squared_array_accum = thrust::raw_pointer_cast(velYSquaredAccum.data());

	thrust::device_vector<float> velXYAccum(particle_count);
	accumulate(vel_xy_array, velYSquaredAccum);
	float* vel_xy_array_accum = thrust::raw_pointer_cast(velXYAccum.data());

	statisticalMomentsKernel2<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(grid_cell_array, vel_x_array_accum,
		vel_y_array_accum, vel_x_squared_array_accum, vel_y_squared_array_accum, vel_xy_array_accum, grid_cell_count);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::resampling()
{
	std::cout << "OccupancyGridMap::resampling" << std::endl;

	CHECK_ERROR(hipDeviceSynchronize());

	float max = static_cast<float>(particle_count + new_born_particle_count);
	thrust::device_vector<float> random_numbers(particle_count);
	thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(particle_count), random_numbers.begin(),
		GPU_LAMBDA(int index)
	{
		thrust::default_random_engine rand_eng;
		thrust::uniform_real_distribution<float> dist(0.0f, max);
		rand_eng.discard(index);
		return dist(rand_eng);
	});
	thrust::sort(random_numbers.begin(), random_numbers.end());

	thrust::device_vector<float> weight_accum(particle_count);
	thrust::device_vector<float> new_born_weight_accum(particle_count);
	accumulate(weight_array, weight_accum);
	accumulate(birth_weight_array, new_born_weight_accum);

	float offset = weight_accum.back();
	thrust::transform(new_born_weight_accum.begin(), new_born_weight_accum.end(), new_born_weight_accum.begin(),
		GPU_LAMBDA(float x)
	{
		return x + offset;
	});

	thrust::device_vector<float> joint_weight_accum(weight_accum.size() + new_born_weight_accum.size());
	joint_weight_accum.insert(joint_weight_accum.end(), weight_accum.begin(), weight_accum.end());
	joint_weight_accum.insert(joint_weight_accum.end(), new_born_weight_accum.begin(), new_born_weight_accum.end());

	thrust::device_vector<int> idx_resampled(particle_count);
	calc_resampled_indeces(joint_weight_accum, random_numbers, idx_resampled);
	int* idx_array_resampled = thrust::raw_pointer_cast(idx_resampled.data());

	resamplingKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, particle_array_next,
		birth_particle_array, idx_array_resampled, particle_count);

	CHECK_ERROR(hipGetLastError());
}
