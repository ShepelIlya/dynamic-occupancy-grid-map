#include "hip/hip_runtime.h"
#include "occupancy_grid_map.h"
#include "cuda_utils.h"
#include "common.h"

#include "kernel/measurement_grid.h"
#include "kernel/init.h"
#include "kernel/predict.h"
#include "kernel/particle_to_grid.h"
#include "kernel/mass_update.h"
#include "kernel/init_new_particles.h"
#include "kernel/update_persistent_particles.h"
#include "kernel/statistical_moments.h"
#include "kernel/resampling.h"

#include <thrust/device_ptr.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>

int OccupancyGridMap::BLOCK_SIZE = 256;

OccupancyGridMap::OccupancyGridMap(const GridParams& params)
	: params(params), 
	  particle_count(params.particle_count),
	  grid_cell_count(static_cast<int>(params.width / params.resolution) * static_cast<int>(params.height / params.resolution)),
	  new_born_particle_count(params.new_born_particle_count)
{
	CHECK_ERROR(hipMallocManaged((void**)&grid_cell_array, grid_cell_count * sizeof(GridCell)));
	CHECK_ERROR(hipMallocManaged((void**)&particle_array, particle_count * sizeof(Particle)));
	CHECK_ERROR(hipMallocManaged((void**)&particle_array_next, particle_count * sizeof(Particle)));
	CHECK_ERROR(hipMalloc((void**)&birth_particle_array, new_born_particle_count * sizeof(Particle)));

	CHECK_ERROR(hipMalloc((void**)&meas_cell_array, grid_cell_count * sizeof(MeasurementCell)));

	CHECK_ERROR(hipMalloc(&weight_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&birth_weight_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&born_masses_array, grid_cell_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_x_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_y_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_x_squared_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_y_squared_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&vel_xy_array, particle_count * sizeof(float)));
	CHECK_ERROR(hipMalloc(&rand_array, particle_count * sizeof(float)));

	initialize();
}

OccupancyGridMap::~OccupancyGridMap()
{
	CHECK_ERROR(hipFree(grid_cell_array));
	CHECK_ERROR(hipFree(particle_array));
	CHECK_ERROR(hipFree(particle_array_next));
	CHECK_ERROR(hipFree(meas_cell_array));

	CHECK_ERROR(hipFree(weight_array));
	CHECK_ERROR(hipFree(birth_weight_array));
	CHECK_ERROR(hipFree(born_masses_array));
	CHECK_ERROR(hipFree(vel_x_array));
	CHECK_ERROR(hipFree(vel_y_array));
	CHECK_ERROR(hipFree(vel_x_squared_array));
	CHECK_ERROR(hipFree(vel_y_squared_array));
	CHECK_ERROR(hipFree(vel_xy_array));
	CHECK_ERROR(hipFree(rand_array));
}

void OccupancyGridMap::initialize()
{
	initParticlesKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, params.width, params.height);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::update(float dt, float* measurements)
{
	updateMeasurementGrid(measurements);

	particlePrediction(dt);
	particleAssignment();
	gridCellOccupancyUpdate();
	updatePersistentParticles();
	initializeNewParticles();
	statisticalMoments();
	resampling();

	CHECK_ERROR(hipMemcpy(particle_array, particle_array_next, sizeof(particle_array_next), hipMemcpyDeviceToDevice));
}

void OccupancyGridMap::updateMeasurementGrid(float* measurements)
{
	float* d_measurements;
	CHECK_ERROR(hipMalloc(&d_measurements, sizeof(measurements)));
	CHECK_ERROR(hipMemcpy(d_measurements, measurements, sizeof(measurements), hipMemcpyHostToDevice));

	const float resolution = 0.2f;
	const float min_range = 0.1f;
	const float max_range = 50.0f;
	const int polar_width = ARRAY_SIZE(measurements);
	const int polar_height = static_cast<int>(max_range / resolution);

	float2* polar_img;
	CHECK_ERROR(hipMalloc(&polar_img, polar_width * polar_height * sizeof(float2)));
	
	dim3 block_dim(32, 32);
	dim3 grid_dim(divUp(polar_width, block_dim.x), divUp(polar_height, block_dim.y));
	
	createPolarGridMapKernel<<<grid_dim, block_dim>>>(polar_img, d_measurements, polar_width, polar_height, resolution,
		min_range, max_range);
	
	CHECK_ERROR(hipGetLastError());
	
	int width = static_cast<int>(params.width / params.resolution);
	int height = static_cast<int>(params.height / params.resolution);
	dim3 cart_grid_dim(divUp(width, block_dim.x), divUp(height, block_dim.y));

	polarToCartesianGridMapKernel<<<cart_grid_dim, block_dim>>>(meas_cell_array, polar_img, width, height,
		polar_width, polar_height);
	
	CHECK_ERROR(hipGetLastError());

	CHECK_ERROR(hipFree(d_measurements));
	CHECK_ERROR(hipFree(polar_img));
}

void OccupancyGridMap::particlePrediction(float dt)
{
	glm::mat4x4 transition_matrix(1, 0, dt, 0, 
								  0, 1, 0, dt, 
								  0, 0, 1, 0, 
								  0, 0, 0, 1);

	thrust::default_random_engine rng;
	thrust::normal_distribution<float> dist_pos(0.0f, params.process_noise_position);
	thrust::normal_distribution<float> dist_vel(0.0f, params.process_noise_velocity);

	glm::vec4 process_noise(dist_pos(rng), dist_pos(rng), dist_vel(rng), dist_vel(rng));

	int width = static_cast<int>(params.width / params.resolution);
	int height = static_cast<int>(params.height / params.resolution);

	predictKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, width, height, params.ps,
		transition_matrix, process_noise);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::particleAssignment()
{
	CHECK_ERROR(hipDeviceSynchronize());
	thrust::device_ptr<Particle> particles(particle_array);
	thrust::sort(particles, particles + particle_count, GPU_LAMBDA(Particle x, Particle y)
	{
		return x.grid_cell_idx < y.grid_cell_idx;
	});

	particleToGridKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, grid_cell_array, weight_array);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::gridCellOccupancyUpdate()
{
	thrust::device_vector<float> weightsAccum(particle_count);
	accumulate(weight_array, weightsAccum);
	float* weight_array_accum = thrust::raw_pointer_cast(weightsAccum.data());

	gridCellPredictionUpdateKernel<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(grid_cell_array, weight_array_accum,
		meas_cell_array, born_masses_array, params.pb);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::updatePersistentParticles()
{
	updatePersistentParticlesKernel1<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, meas_cell_array,
		weight_array);

	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());

	thrust::device_vector<float> weightsAccum(particle_count);
	accumulate(weight_array, weightsAccum);
	float* weight_array_accum = thrust::raw_pointer_cast(weightsAccum.data());

	updatePersistentParticlesKernel2<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(grid_cell_array,
		weight_array_accum);

	CHECK_ERROR(hipGetLastError());

	updatePersistentParticlesKernel3<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, meas_cell_array,
		grid_cell_array, weight_array);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::initializeNewParticles()
{
	thrust::device_vector<float> particleOrdersAccum(grid_cell_count);
	accumulate(born_masses_array, particleOrdersAccum);
	float* particle_orders_array_accum = thrust::raw_pointer_cast(particleOrdersAccum.data());

	normalize_particle_orders(particle_orders_array_accum, params.new_born_particle_count);

	initNewParticlesKernel1<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, grid_cell_array,
		meas_cell_array, weight_array, born_masses_array, birth_particle_array, particle_orders_array_accum);

	CHECK_ERROR(hipGetLastError());

	initNewParticlesKernel2<<<divUp(new_born_particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(birth_particle_array,
		grid_cell_array, birth_weight_array, static_cast<int>(params.width / params.resolution));

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::statisticalMoments()
{
	statisticalMomentsKernel1<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, weight_array,
		vel_x_array, vel_y_array, vel_x_squared_array, vel_y_squared_array, vel_xy_array);

	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());

	thrust::device_vector<float> velXAccum(particle_count);
	accumulate(vel_x_array, velXAccum);
	float* vel_x_array_accum = thrust::raw_pointer_cast(velXAccum.data());

	thrust::device_vector<float> velYAccum(particle_count);
	accumulate(vel_y_array, velYAccum);
	float* vel_y_array_accum = thrust::raw_pointer_cast(velYAccum.data());

	thrust::device_vector<float> velXSquaredAccum(particle_count);
	accumulate(vel_x_squared_array, velXSquaredAccum);
	float* vel_x_squared_array_accum = thrust::raw_pointer_cast(velXSquaredAccum.data());

	thrust::device_vector<float> velYSquaredAccum(particle_count);
	accumulate(vel_y_squared_array, velYSquaredAccum);
	float* vel_y_squared_array_accum = thrust::raw_pointer_cast(velYSquaredAccum.data());

	thrust::device_vector<float> velXYAccum(particle_count);
	accumulate(vel_xy_array, velYSquaredAccum);
	float* vel_xy_array_accum = thrust::raw_pointer_cast(velXYAccum.data());

	statisticalMomentsKernel2<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(grid_cell_array, vel_x_array_accum,
		vel_y_array_accum, vel_x_squared_array_accum, vel_y_squared_array_accum, vel_xy_array_accum);

	CHECK_ERROR(hipGetLastError());
}

void OccupancyGridMap::resampling()
{
	resamplingKernel<<<divUp(particle_count, BLOCK_SIZE), BLOCK_SIZE>>>(particle_array, particle_array_next,
		birth_particle_array, rand_array, nullptr/*idx_array_resampled*/);

	CHECK_ERROR(hipGetLastError());
}
