#include "OccupancyGridMap.h"
#include "OccupancyGridKernel.h"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/random.h>
#include <Eigen/Dense>

#include "hip/hip_runtime.h"

#define ARRAY_SIZE(x) sizeof(x) / sizeof(x[0])

OccupancyGridMap::OccupancyGridMap(const GridParams& params)
	: params(params)
{
	int c = (params.width / params.resolution) * (params.height / params.resolution);

	hipMallocManaged((void**)&grid_cell_array, c * sizeof(GridCell));
	hipMallocManaged((void**)&particle_array, params.v * sizeof(Particle));
	hipMallocManaged(&weight_array, params.v * sizeof(float));
	hipMallocManaged(&birth_weight_array, params.v * sizeof(float));
	hipMallocManaged(&meas_cell_array, c * sizeof(MeasurementCell));

	hipMalloc(&born_masses_array, c * sizeof(float));
	hipMalloc(&vel_x_array, params.v * sizeof(float));
	hipMalloc(&vel_y_array, params.v * sizeof(float));
	hipMalloc(&vel_x_squared_array, params.v * sizeof(float));
	hipMalloc(&vel_y_squared_array, params.v * sizeof(float));
	hipMalloc(&vel_xy_array, params.v * sizeof(float));
	hipMalloc(&rand_array, params.v * sizeof(float));

	initializationKernel/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, params.width, params.height,
		ARRAY_SIZE(particle_array));
}

OccupancyGridMap::~OccupancyGridMap()
{
	hipFree(grid_cell_array);
	hipFree(particle_array);
	hipFree(weight_array);
	hipFree(birth_weight_array);
	hipFree(meas_cell_array);

	hipFree(born_masses_array);
	hipFree(vel_x_array);
	hipFree(vel_y_array);
	hipFree(vel_x_squared_array);
	hipFree(vel_y_squared_array);
	hipFree(vel_xy_array);
	hipFree(rand_array);
}

void OccupancyGridMap::update(float t)
{
	particlePrediction(t);

	particleAssignment();

	gridCellOccupancyUpdate();

	updatePersistentParticles();

	initializeNewParticles();

	statisticalMoments();

	resampling();
}

void OccupancyGridMap::particlePrediction(float t)
{
	Eigen::Matrix4f transitionMatrix;
	transitionMatrix << 1, 0, t, 0,
						0, 1, 0, t,
						0, 0, 1, 0,
						0, 0, 0, 1;

	thrust::default_random_engine rng;
	thrust::normal_distribution<float> distPos(0.0f, params.processNoisePosition);
	thrust::normal_distribution<float> distVel(0.0f, params.processNoiseVelocity);

	Eigen::Vector4f zeta;
	zeta << distPos(rng), distPos(rng), distVel(rng), distVel(rng);
	
	predictKernel/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, pS, transitionMatrix, zeta);
}

void OccupancyGridMap::particleAssignment()
{
	struct SortParticles
	{
		__host__ __device__ bool operator()(Particle x, Particle y)
		{
			return x.gridCellIdx < y.gridCellIdx;
		}
	};
	hipDeviceSynchronize();
	thrust::device_ptr<Particle> particles = thrust::device_pointer_cast(particle_array);
	thrust::sort(particles, particles + ARRAY_SIZE(particle_array), SortParticles());

	particleToGridKernel/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, grid_cell_array, weight_array);
}

void OccupancyGridMap::gridCellOccupancyUpdate()
{
	thrust::device_vector<float> weightsAccum = accumulate(weight_array);
	float* weight_array_accum = thrust::raw_pointer_cast(&weightsAccum[0]);
	gridCellPredictionUpdateKernel/*<<<(gridSize + 256 - 1) / 256, 256>>>*/(grid_cell_array, weight_array_accum, meas_cell_array, born_masses_array, pB);
}

void OccupancyGridMap::updatePersistentParticles()
{
	updatePersistentParticlesKernel1/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, meas_cell_array, weight_array);
	hipDeviceSynchronize();
	thrust::device_vector<float> weightsAccum = accumulate(weight_array);
	float* weight_array_accum = thrust::raw_pointer_cast(&weightsAccum[0]);

	updatePersistentParticlesKernel2/*<<<(gridSize + 256 - 1) / 256, 256>>>*/(grid_cell_array, weight_array_accum);
	updatePersistentParticlesKernel3/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, grid_cell_array, weight_array);
}

void OccupancyGridMap::initializeNewParticles()
{
	thrust::device_vector<float> particleOrdersAccum = accumulate(born_masses_array);
	float* particle_orders_array_accum = thrust::raw_pointer_cast(&particleOrdersAccum[0]);
	normalize_particle_orders(particle_orders_array_accum, vB);
	initNewParticlesKernel1/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, grid_cell_array, meas_cell_array, 
		weight_array, born_masses_array, birth_particle_array, particle_orders_array_accum);
	initNewParticlesKernel2/*<<<(birtParticlesSize + 256 - 1) / 256, 256>>>*/(birth_particle_array, grid_cell_array);
}

void OccupancyGridMap::statisticalMoments()
{
	statisticalMomentsKernel1/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, weight_array, vel_x_array,
		vel_y_array, vel_x_squared_array, vel_y_squared_array, vel_xy_array);

	hipDeviceSynchronize();

	thrust::device_vector<float> velXAccum = accumulate(vel_x_array);
	thrust::device_vector<float> velYAccum = accumulate(vel_y_array);
	thrust::device_vector<float> velXSquaredAccum = accumulate(vel_x_squared_array);
	thrust::device_vector<float> velYSquaredAccum = accumulate(vel_y_squared_array);
	thrust::device_vector<float> velXYAccum = accumulate(vel_xy_array);

	float* vel_x_array_accum = thrust::raw_pointer_cast(&velXAccum[0]);
	float* vel_y_array_accum = thrust::raw_pointer_cast(&velYAccum[0]);
	float* vel_x_squared_array_accum = thrust::raw_pointer_cast(&velXSquaredAccum[0]);
	float* vel_y_squared_array_accum = thrust::raw_pointer_cast(&velYSquaredAccum[0]);
	float* vel_xy_array_accum = thrust::raw_pointer_cast(&velXYAccum[0]);

	statisticalMomentsKernel2/*<<<(gridSize + 256 - 1) / 256, 256>>>*/(grid_cell_array, vel_x_array_accum, vel_y_array_accum, 
		vel_x_squared_array_accum, vel_y_squared_array_accum, vel_xy_array_accum);
}

void OccupancyGridMap::resampling()
{
	resamplingKernel/*<<<(ARRAY_SIZE(particle_array) + 256 - 1) / 256, 256>>>*/(particle_array, particle_array/*_next*/, birth_particle_array,
		rand_array, nullptr/*idx_array_resampled*/);
}
