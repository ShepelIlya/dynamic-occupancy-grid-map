#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael K�sel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include "kernel/init.h"
#include "common.h"
#include "cuda_utils.h"

#include <thrust/random.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void initParticlesKernel(Particle* particle_array, int grid_size, int particle_count)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
	{
		unsigned int seed = hash(i);
		thrust::default_random_engine rng(seed);
		thrust::uniform_int_distribution<int> dist_idx(0, grid_size * grid_size);
		thrust::normal_distribution<double> dist_vel(0.0f, 12.0f);

		int index = dist_idx(rng);

		double x = index % grid_size;
		double y = index / grid_size;

		particle_array[i].weight = 1.0 / static_cast<double>(particle_count);
		particle_array[i].state = glm::vec4(x, y, dist_vel(rng), dist_vel(rng));

		//printf("w: %f, x: %f, y: %f, vx: %f, vy: %f\n", particle_array[i].weight, particle_array[i].state[0], particle_array[i].state[1],
		//	particle_array[i].state[2], particle_array[i].state[3]);
	}
}

__global__ void initGridCellsKernel(GridCell* grid_cell_array, MeasurementCell* meas_cell_array, int grid_size, int cell_count)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < cell_count; i += blockDim.x * gridDim.x)
	{
		const int x = i % grid_size;
		const int y = i / grid_size;
		grid_cell_array[i].pos = make_int2(x, y);
		grid_cell_array[i].free_mass = 0.0f;
		grid_cell_array[i].occ_mass = 0.0f;
		//grid_cell_array[i].start_idx = -1;
		//grid_cell_array[i].end_idx = -1;

		meas_cell_array[i].occ_mass = 0.0f;
		meas_cell_array[i].free_mass = 0.0f;
		meas_cell_array[i].likelihood = 1.0f;
		meas_cell_array[i].p_A = 1.0f;
	}
}

__global__ void reinitGridParticleIndices(GridCell* grid_cell_array, int cell_count)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < cell_count; i += blockDim.x * gridDim.x)
	{
		grid_cell_array[i].start_idx = -1;
		grid_cell_array[i].end_idx = -1;
	}
}
