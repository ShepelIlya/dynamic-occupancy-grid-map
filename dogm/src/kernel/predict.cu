#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael K�sel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include "kernel/predict.h"
#include "cuda_utils.h"
#include "common.h"

#include <thrust/random.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void predictKernel(Particle* particle_array, int grid_size, double p_S, const glm::mat4x4 transition_matrix,
	const glm::vec4 process_noise, int particle_count)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
	{
		particle_array[i].state = transition_matrix * particle_array[i].state + process_noise;
		particle_array[i].weight = p_S * particle_array[i].weight;

		double x = particle_array[i].state[0];
		double y = particle_array[i].state[1];

		if ((x > grid_size - 1 || x < 0) || (y > grid_size - 1 || y < 0))
		{
			unsigned int seed = hash(i);
			thrust::default_random_engine rng(seed);
			thrust::uniform_int_distribution<int> dist_idx(0, grid_size * grid_size);
			thrust::normal_distribution<double> dist_vel(0.0f, 12.0);

			const int index = dist_idx(rng);

			x = index % grid_size;
			y = index / grid_size;

			particle_array[i].state = glm::vec4(x, y, dist_vel(rng), dist_vel(rng));
		}

		int pos_x = clamp(static_cast<int>(x), 0, grid_size - 1);
		int pos_y = clamp(static_cast<int>(y), 0, grid_size - 1);
		particle_array[i].grid_cell_idx = pos_x + grid_size * pos_y;

		//printf("X: %d, Y: %d, Cell index: %d\n", pos_x, pos_y, (pos_x + grid_size * pos_y));
	}
}
